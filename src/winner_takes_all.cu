#include "hip/hip_runtime.h"
/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "internal.h"

#include <hip/hip_runtime.h>

#include "device_utility.h"
#include "host_utility.h"

namespace sgm
{
namespace
{

static constexpr unsigned int WARPS_PER_BLOCK = 8u;
static constexpr unsigned int BLOCK_SIZE = WARPS_PER_BLOCK * WARP_SIZE;

__device__ inline uint32_t pack_cost_index(uint32_t cost, uint32_t index)
{
	union {
		uint32_t uint32;
		ushort2 uint16x2;
	} u;
	u.uint16x2.x = static_cast<uint16_t>(index);
	u.uint16x2.y = static_cast<uint16_t>(cost);
	return u.uint32;
}

__device__ uint32_t unpack_cost(uint32_t packed)
{
	return packed >> 16;
}

__device__ int unpack_index(uint32_t packed)
{
	return packed & 0xffffu;
}

using ComputeDisparity = uint32_t(*)(uint32_t, uint32_t, uint16_t*);

__device__ inline uint32_t compute_disparity_normal(uint32_t disp, uint32_t cost = 0, uint16_t* smem = nullptr)
{
	return disp;
}

template <size_t MAX_DISPARITY>
__device__ inline uint32_t compute_disparity_subpixel(uint32_t disp, uint32_t cost, uint16_t* smem)
{
	int subp = disp;
	subp <<= sgm::StereoSGM::SUBPIXEL_SHIFT;
	if (disp > 0 && disp < MAX_DISPARITY - 1) {
		const int left = smem[disp - 1];
		const int right = smem[disp + 1];
		const int numer = left - right;
		const int denom = left - 2 * cost + right;
		subp += ((numer << sgm::StereoSGM::SUBPIXEL_SHIFT) + denom) / (2 * denom);
	}
	return subp;
}


template <unsigned int MAX_DISPARITY, unsigned int NUM_PATHS, ComputeDisparity compute_disparity = compute_disparity_normal>
__global__ void winner_takes_all_kernel(
	output_type *left_dest,
	output_type *right_dest,
	const cost_type *src,
	int width,
	int height,
	int pitch,
	float uniqueness)
{
	static const unsigned int ACCUMULATION_PER_THREAD = 16u;
	static const unsigned int REDUCTION_PER_THREAD = MAX_DISPARITY / WARP_SIZE;
	static const unsigned int ACCUMULATION_INTERVAL = ACCUMULATION_PER_THREAD / REDUCTION_PER_THREAD;
	static const unsigned int UNROLL_DEPTH = 
		(REDUCTION_PER_THREAD > ACCUMULATION_INTERVAL)
			? REDUCTION_PER_THREAD
			: ACCUMULATION_INTERVAL;

	const size_t cost_step = static_cast<size_t>(MAX_DISPARITY) * width * height;
	const unsigned int warp_id = threadIdx.x / WARP_SIZE;
	const unsigned int lane_id = threadIdx.x % WARP_SIZE;

	const unsigned int y = blockIdx.x * WARPS_PER_BLOCK + warp_id;
	src += y * MAX_DISPARITY * width;
	left_dest  += y * pitch;
	right_dest += y * pitch;

	if(y >= height){
		return;
	}

	__shared__ uint16_t smem_cost_sum[WARPS_PER_BLOCK][ACCUMULATION_INTERVAL][MAX_DISPARITY];

	uint32_t right_best[REDUCTION_PER_THREAD];
	for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
		right_best[i] = 0xffffffffu;
	}

	for(unsigned int x0 = 0; x0 < width; x0 += UNROLL_DEPTH){
#pragma unroll
		for(unsigned int x1 = 0; x1 < UNROLL_DEPTH; ++x1){
			if(x1 % ACCUMULATION_INTERVAL == 0){
				const unsigned int k = lane_id * ACCUMULATION_PER_THREAD;
				const unsigned int k_hi = k / MAX_DISPARITY;
				const unsigned int k_lo = k % MAX_DISPARITY;
				const unsigned int x = x0 + x1 + k_hi;
				if(x < width){
					const unsigned int offset = x * MAX_DISPARITY + k_lo;
					uint32_t sum[ACCUMULATION_PER_THREAD];
					for(unsigned int i = 0; i < ACCUMULATION_PER_THREAD; ++i){
						sum[i] = 0;
					}
					for(unsigned int p = 0; p < NUM_PATHS; ++p){
						uint32_t load_buffer[ACCUMULATION_PER_THREAD];
						load_uint8_vector<ACCUMULATION_PER_THREAD>(
							load_buffer, &src[p * cost_step + offset]);
						for(unsigned int i = 0; i < ACCUMULATION_PER_THREAD; ++i){
							sum[i] += load_buffer[i];
						}
					}
					store_uint16_vector<ACCUMULATION_PER_THREAD>(
						&smem_cost_sum[warp_id][k_hi][k_lo], sum);
				}
#if CUDA_VERSION >= 9000
				__syncwarp();
#else
				__threadfence_block();
#endif
			}
			const unsigned int x = x0 + x1;
			if(x < width){
				// Load sum of costs
				const unsigned int smem_x = x1 % ACCUMULATION_INTERVAL;
				const unsigned int k0 = lane_id * REDUCTION_PER_THREAD;
				uint32_t local_cost_sum[REDUCTION_PER_THREAD];
				load_uint16_vector<REDUCTION_PER_THREAD>(
					local_cost_sum, &smem_cost_sum[warp_id][smem_x][k0]);
				// Pack sum of costs and dispairty
				uint32_t local_packed_cost[REDUCTION_PER_THREAD];
				for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
					local_packed_cost[i] = pack_cost_index(local_cost_sum[i], k0 + i);
				}
				// Update left
				uint32_t best = 0xffffffffu;
				for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
					best = min(best, local_packed_cost[i]);
				}
				best = subgroup_min<WARP_SIZE>(best, 0xffffffffu);
				// Update right
#pragma unroll
				for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
					const unsigned int k = lane_id * REDUCTION_PER_THREAD + i;
					const int p = static_cast<int>(((x - k) & ~(MAX_DISPARITY - 1)) + k);
					const unsigned int d = static_cast<unsigned int>(x - p);
#if CUDA_VERSION >= 9000
					const uint32_t recv = __shfl_sync(0xffffffffu,
						local_packed_cost[(REDUCTION_PER_THREAD - i + x1) % REDUCTION_PER_THREAD],
						d / REDUCTION_PER_THREAD,
						WARP_SIZE);
#else
					const uint32_t recv = __shfl(
						local_packed_cost[(REDUCTION_PER_THREAD - i + x1) % REDUCTION_PER_THREAD],
						d / REDUCTION_PER_THREAD,
						WARP_SIZE);
#endif
					right_best[i] = min(right_best[i], recv);
					if(d == MAX_DISPARITY - 1){
						if(0 <= p){
							right_dest[p] = compute_disparity_normal(unpack_index(right_best[i]));
						}
						right_best[i] = 0xffffffffu;
					}
				}
				// Resume updating left to avoid execution dependency
				const uint32_t bestCost = unpack_cost(best);
				const int bestDisp = unpack_index(best);
				bool uniq = true;
				for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
					const uint32_t x = local_packed_cost[i];
					const bool uniq1 = unpack_cost(x) * uniqueness >= bestCost;
					const bool uniq2 = abs(unpack_index(x) - bestDisp) <= 1;
					uniq &= uniq1 || uniq2;
				}
				uniq = subgroup_and<WARP_SIZE>(uniq, 0xffffffffu);
				if(lane_id == 0){
					left_dest[x] = uniq ? compute_disparity(bestDisp, bestCost, smem_cost_sum[warp_id][smem_x]) : INVALID_DISP;
				}
			}
		}
	}
	for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
		const unsigned int k = lane_id * REDUCTION_PER_THREAD + i;
		const int p = static_cast<int>(((width - k) & ~(MAX_DISPARITY - 1)) + k);
		if(0 <= p && p < width){
			right_dest[p] = compute_disparity_normal(unpack_index(right_best[i]));
		}
	}
}

} // namespace

namespace details
{

template <int MAX_DISPARITY>
void winner_takes_all_(const DeviceImage& src, DeviceImage& dstL, DeviceImage& dstR,
	float uniqueness, bool subpixel, PathType path_type)
{
	const int width = dstL.cols;
	const int height = dstL.rows;
	const int pitch = dstL.step;

	const int gdim = divUp(height, WARPS_PER_BLOCK);
	const int bdim = BLOCK_SIZE;

	const cost_type* cost = src.ptr<cost_type>();
	output_type* dispL = dstL.ptr<output_type>();
	output_type* dispR = dstR.ptr<output_type>();

	switch (path_type) {
		case PathType::SCAN_4PATH: if (subpixel) {
			winner_takes_all_kernel<MAX_DISPARITY, 4, compute_disparity_subpixel<MAX_DISPARITY>><<<gdim, bdim>>>(
				dispL, dispR, cost, width, height, pitch, uniqueness);
		} else {
			winner_takes_all_kernel<MAX_DISPARITY, 4, compute_disparity_normal><<<gdim, bdim>>>(
				dispL, dispR, cost, width, height, pitch, uniqueness);
		} break;

		case PathType::SCAN_8PATH: if (subpixel) {
			winner_takes_all_kernel<MAX_DISPARITY, 8, compute_disparity_subpixel<MAX_DISPARITY>><<<gdim, bdim>>>(
				dispL, dispR, cost, width, height, pitch, uniqueness);
		} else {
			winner_takes_all_kernel<MAX_DISPARITY, 8, compute_disparity_normal><<<gdim, bdim>>>(
				dispL, dispR, cost, width, height, pitch, uniqueness);
		} break;

		case PathType::SCAN_16PATH: if (subpixel) {
			winner_takes_all_kernel<MAX_DISPARITY, 16, compute_disparity_subpixel<MAX_DISPARITY>><<<gdim, bdim>>>(
				dispL, dispR, cost, width, height, pitch, uniqueness);
		} else {
			winner_takes_all_kernel<MAX_DISPARITY, 16, compute_disparity_normal><<<gdim, bdim>>>(
				dispL, dispR, cost, width, height, pitch, uniqueness);
		} break;

		default: assert(("unimplemented", false));
	}

	CUDA_CHECK(hipGetLastError());
}

void winner_takes_all(const DeviceImage& src, DeviceImage& dstL, DeviceImage& dstR,
	int disp_size, float uniqueness, bool subpixel, PathType path_type)
{
	if (disp_size == 64) {
		winner_takes_all_<64>(src, dstL, dstR, uniqueness, subpixel, path_type);
	}
	else if (disp_size == 128) {
		winner_takes_all_<128>(src, dstL, dstR, uniqueness, subpixel, path_type);
	}
	else if (disp_size == 256) {
		winner_takes_all_<256>(src, dstL, dstR, uniqueness, subpixel, path_type);
	}
}

} // namespace details
} // namespace sgm
